#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include <stdio.h>
#include <pthread.h>
#include <time.h>
#include "gpu-new-forward.h"
#include "hip/hip_fp16.h"

#define TILE_WIDTH 16
#define K_COMMON 7
#define C_LAYER1 1
#define M_LAYER1 4
#define C_LAYER2 4
#define M_LAYER2 16
#define TILE_WIDTH_1D 512 //GPU implementation
#define NUM_PTHREADS 32


__constant__ half k_layer1[K_COMMON * K_COMMON * C_LAYER1 * M_LAYER1];
__constant__ half k_layer2[K_COMMON * K_COMMON * C_LAYER2 * M_LAYER2];


__global__ void conv_forward_kernel_layer1(float *y, const float * __restrict__ x, const int B, const int M, const int C, const int H, const int W, const int K)
{
    /*
    Modify this function to implement the forward pass described in Chapter 16.
    We have added an additional dimension to the tensors to support an entire mini-batch
    The goal here is to be correct AND fast.

    Function paramter definitions:
    y - output
    x - input
    k - kernel
    B - batch_size (number of images in x)
    M - number of output feature maps
    C - number of input feature maps
    H - input height dimension
    W - input width dimension
    K - kernel height and width (K x K)
    */

    const int H_out = H - K + 1;
    const int W_out = W - K + 1;
    // (void)H_out; // silence declared but never referenced warning. remove this line when you start working
    // (void)W_out; // silence declared but never referenced warning. remove this line when you start working

    // We have some nice #defs for you below to simplify indexing. Feel free to use them, or create your own.
    // An example use of these macros:
    // float a = y4d(0,0,0,0)
    // y4d(0,0,0,0) = a

#define y4d(i3, i2, i1, i0) y[(i3) * (M * H_out * W_out) + (i2) * (H_out * W_out) + (i1) * (W_out) + i0]
#define x4d(i3, i2, i1, i0) x[(i3) * (C * H * W) + (i2) * (H * W) + (i1) * (W) + i0]
#define k4d(i3, i2, i1, i0) k_layer1[(i3) * (C * K * K) + (i2) * (K * K) + (i1) * (K) + i0]
#define loc_mem(i2, i1, i0) loc_mem1[(i2) * ((TILE_WIDTH+K_COMMON-1) * (TILE_WIDTH+K_COMMON-1)) + (i1) * (TILE_WIDTH+K_COMMON-1) + (i0)]

    // Insert your GPU convolution kernel code here
    int W_grid = ceil((1.0 * W_out)/TILE_WIDTH);
    int H_grid = ceil((1.0 * H_out)/TILE_WIDTH);

    int b,m,h,w;
    b = blockIdx.x;
    m = blockIdx.y;
    h = (blockIdx.z/W_grid)*TILE_WIDTH + threadIdx.y;
    w = (blockIdx.z % W_grid)*TILE_WIDTH + threadIdx.x;

    __shared__ half loc_mem1[(TILE_WIDTH + K_COMMON - 1) * (TILE_WIDTH + K_COMMON - 1)];

    loc_mem(0, threadIdx.y, threadIdx.x) = __float2half(x4d(b, 0, h, w));
    loc_mem(0, threadIdx.y + (K-1), threadIdx.x) = __float2half(x4d(b, 0, h + (K - 1), w));
    loc_mem(0, threadIdx.y, threadIdx.x + (K - 1)) = __float2half(x4d(b, 0, h, w + (K - 1)));
    loc_mem(0, threadIdx.y + (K - 1), threadIdx.x + (K - 1)) = __float2half(x4d(b, 0, h + (K - 1), w + (K - 1)));

    __syncthreads();

    half acc = 0.0f;
    #pragma unroll 7
    for (int p = 0; p < K; p++) {
        #pragma unroll 7
        for (int q = 0; q < K; q++) {
            acc = __hadd(acc, __hmul(loc_mem(0, threadIdx.y+p, threadIdx.x+q), k4d(m, 0, p, q))); // C = C_LAYER1 - 1
        }
    }
    y4d(b, m, h, w) = __half2float(acc);

#undef y4d
#undef x4d
#undef k4d
#undef loc_mem
}

__global__ void conv_forward_kernel_layer2(float *y, const float * __restrict__ x, const int B, const int M, const int C, const int H, const int W, const int K)
{
    /*
    Modify this function to implement the forward pass described in Chapter 16.
    We have added an additional dimension to the tensors to support an entire mini-batch
    The goal here is to be correct AND fast.

    Function paramter definitions:
    y - output
    x - input
    k - kernel
    B - batch_size (number of images in x)
    M - number of output feature maps
    C - number of input feature maps
    H - input height dimension
    W - input width dimension
    K - kernel height and width (K x K)
    */

    const int H_out = H - K + 1;
    const int W_out = W - K + 1;
    // (void)H_out; // silence declared but never referenced warning. remove this line when you start working
    // (void)W_out; // silence declared but never referenced warning. remove this line when you start working

    // We have some nice #defs for you below to simplify indexing. Feel free to use them, or create your own.
    // An example use of these macros:
    // float a = y4d(0,0,0,0)
    // y4d(0,0,0,0) = a

#define y4d(i3, i2, i1, i0) y[(i3) * (M * H_out * W_out) + (i2) * (H_out * W_out) + (i1) * (W_out) + i0]
#define x4d(i3, i2, i1, i0) x[(i3) * (C * H * W) + (i2) * (H * W) + (i1) * (W) + i0]
#define k4d(i3, i2, i1, i0) k_layer2[(i3) * (C * K * K) + (i2) * (K * K) + (i1) * (K) + i0]
#define loc_mem(i2, i1, i0) loc_mem2[(i2) * ((TILE_WIDTH+K-1) * (TILE_WIDTH+K-1)) + (i1) * (TILE_WIDTH+K-1) + (i0)]

    // Insert your GPU convolution kernel code here
    int W_grid = ceil((1.0 * W_out)/TILE_WIDTH);
    int H_grid = ceil((1.0 * H_out)/TILE_WIDTH);

    int b,m,h,w;
    b = blockIdx.x;
    m = blockIdx.y;
    h = (blockIdx.z/W_grid)*TILE_WIDTH + threadIdx.y;
    w = (blockIdx.z % W_grid)*TILE_WIDTH + threadIdx.x;

    __shared__ half loc_mem2[C_LAYER2 * (TILE_WIDTH + K_COMMON - 1) * (TILE_WIDTH + K_COMMON - 1)]; // set size on lauch

    #pragma unroll 4 //6 //7
    for(int c = 0; c < C; c++) {
      loc_mem(c, threadIdx.y, threadIdx.x) = __float2half(x4d(b, c, h, w));
      loc_mem(c, threadIdx.y + (K-1), threadIdx.x) = __float2half(x4d(b, c, h + (K - 1), w));
      loc_mem(c, threadIdx.y, threadIdx.x + (K - 1)) = __float2half(x4d(b, c, h, w + (K - 1)));
      loc_mem(c, threadIdx.y + (K - 1), threadIdx.x + (K - 1)) = __float2half(x4d(b, c, h + (K - 1), w + (K - 1)));
    }

    __syncthreads();

    if (h < H_out && w < W_out) {
        half acc = 0.0f;
        #pragma unroll 4 //6
        for (int c = 0; c < C; c++) {
            #pragma unroll 7
            for (int p = 0; p < K; p++) {
                #pragma unroll 7
                for (int q = 0; q < K; q++) {
                    acc = __hadd(acc, __hmul(loc_mem(c, threadIdx.y+p, threadIdx.x+q), k4d(m, c, p, q)));
                }
            }
        }
        y4d(b, m, h, w) = __half2float(acc);
    }

#undef y4d
#undef x4d
#undef k4d
#undef loc_mem
}

struct args {
  const float *f_arr_k;
  half *h_arr_k;
  int len_k;
  int start_idx_k;
};

struct args2 {
  float *f_arr;
  half * h_arr;
  int len;
  int start_idx;
};

struct args_pinned {
  const float *copy_from_address;
  float *copy_to_address;
  int len;
  int start_idx;
  int end_idx;
};

__host__ void *pinned_transfer(void *args_pinned_list) {
  int len = ((struct args_pinned *)args_pinned_list)->len;
  int start_idx = ((struct args_pinned *)args_pinned_list)->start_idx;
  int end_idx = ((struct args_pinned *)args_pinned_list)->end_idx;
  float *copy_to_address = ((struct args_pinned *)args_pinned_list)->copy_to_address;
  const float *copy_from_address = ((struct args_pinned *)args_pinned_list)->copy_from_address;

  #pragma unroll 7
  for (int i = start_idx; i < end_idx; i++) {
    if (i >= len) 
      break;
    copy_to_address[i] = copy_from_address[i];
  }
  return NULL;
}

__host__ void * f2h_host(void *arg_list){

  int len_k = ((struct args *)arg_list)->len_k;
  #pragma unroll 7
  for(int i = 0; i < ceil((1.0*len_k)/(NUM_PTHREADS)); i++){
    int idx  = ((struct args *)arg_list)->start_idx_k + i;
    if(idx >= len_k)
      break;
    (((struct args *)arg_list)->h_arr_k)[idx] = __float2half((((struct args *)arg_list)->f_arr_k)[idx]);
  }
  return NULL;
}

__host__ void * h2f_host(void *arg_list){ // bypasses const by converting from void *
  int len = ((struct args2 *)arg_list)->len;
  #pragma unroll 7
  for(int i = 0; i < ceil((1.0*len)/(NUM_PTHREADS)); i++){
    int idx  = ((struct args2 *)arg_list)->start_idx + i;
    if(idx >= len)
      break;
    (((struct args2 *)arg_list)->f_arr)[idx] = __float2half((((struct args2 *)arg_list)->h_arr)[idx]);
  }
  return NULL;
}

__host__ void GPUInterface::conv_forward_gpu6(float *host_y, const float *host_x, const float *host_k, const int B,
                                            const int M, const int C, const int H, const int W, const int K)
{   
    int small_B = 1000;
    hipStream_t stream0, stream1;
    hipStreamCreate(&stream0);
    hipStreamCreate(&stream1);

    float *device_y0, *device_y1;
    float *device_x0, *device_x1;
    float *host_pinned_x;
    float *host_pinned_y;

    const int H_out = H - K + 1;
    const int W_out = W - K + 1;

    int x_len = (W * H * C * B);
    int y_len = (H_out * W_out * M * B);
    int k_len = (K * K * M * C);

    //Streams
    int x_len_stream = (W * H * C * small_B);
    int y_len_stream = (H_out * W_out * M * small_B);

    half *host_k16 = (half *)malloc(k_len * sizeof(half));
    // Allocate memory and copy over the relevant data structures to the GPU

    pthread_t tids[NUM_PTHREADS];
    struct args arg_list[NUM_PTHREADS];

    #pragma unroll NUM_PTHREADS
    for (unsigned int i = 0; i <  NUM_PTHREADS; i++) {
        arg_list[i].len_k = k_len;
        arg_list[i].f_arr_k = host_k;
        arg_list[i].h_arr_k = host_k16;
        arg_list[i].start_idx_k = i*ceil((1.0*k_len)/NUM_PTHREADS);
        pthread_create(tids + i, NULL, f2h_host, (void *)(arg_list + i));
    }

    #pragma unroll NUM_PTHREADS
    for (unsigned int i = 0; i <  NUM_PTHREADS; i++) {
      pthread_join(tids[i], NULL);
    }

    hipMalloc((void **) &device_y0, (y_len_stream * sizeof(float)));
    hipMalloc((void **) &device_y1, (y_len_stream * sizeof(float)));
    hipMalloc((void **) &device_x0, (x_len_stream * sizeof(float)));
    hipMalloc((void **) &device_x1, (x_len_stream * sizeof(float)));

    int W_grid = ceil((1.0 * W_out)/TILE_WIDTH);
    int H_grid = ceil((1.0 * H_out)/TILE_WIDTH);
    int Z = W_grid * H_grid;

    // put device_k into constant memory
    if (C == 1) { hipMemcpyToSymbol(HIP_SYMBOL(k_layer1), host_k16, K * K * C_LAYER1 * M_LAYER1 * sizeof(half)); }
    else { hipMemcpyToSymbol(HIP_SYMBOL(k_layer2), host_k16, K * K * C_LAYER2 * M_LAYER2 * sizeof(half)); }

    //Use Pinned Memory
    hipHostAlloc((void **)&host_pinned_x, x_len * sizeof(float), hipHostMallocDefault);
    hipHostAlloc((void **)&host_pinned_y, y_len * sizeof(float), hipHostMallocDefault);

    struct args_pinned args_pinned_list[NUM_PTHREADS];

    #pragma unroll NUM_PTHREADS
    for (unsigned int i = 0; i <  NUM_PTHREADS; i++) {
        args_pinned_list[i].len = x_len;
        args_pinned_list[i].copy_from_address = host_x;
        args_pinned_list[i].copy_to_address = host_pinned_x;
        args_pinned_list[i].start_idx = i*ceil((1.0*x_len)/NUM_PTHREADS);
        args_pinned_list[i].end_idx = (i+1)*ceil((1.0*x_len)/NUM_PTHREADS);
        pthread_create(tids + i, NULL, pinned_transfer, (void *)(args_pinned_list + i));
    }

    #pragma unroll NUM_PTHREADS
    for (unsigned int i = 0; i <  NUM_PTHREADS; i++) {
      pthread_join(tids[i], NULL);
    }

    dim3 DimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    dim3 DimGrid(small_B, M, Z);

    //Use streams to launch and run kernels 
    for (int i = 0; i < B; i = i + small_B * 2) {
        hipMemcpyAsync(device_x0, host_pinned_x + i * (x_len_stream/small_B), x_len_stream * sizeof(float), hipMemcpyHostToDevice, stream0);
        hipMemcpyAsync(device_x1, host_pinned_x + i * (x_len_stream/small_B) + x_len_stream, x_len_stream * sizeof(float), hipMemcpyHostToDevice, stream1);
        if (C == 1) { 
          conv_forward_kernel_layer1<<<DimGrid, DimBlock, 0, stream0>>> (device_y0, device_x0, B, M, C, H, W, K);
          conv_forward_kernel_layer1<<<DimGrid, DimBlock, 0, stream1>>> (device_y1, device_x1, B, M, C, H, W, K); 
        }
        else { 
          conv_forward_kernel_layer2<<<DimGrid, DimBlock, 0, stream0>>> (device_y0, device_x0, B, M, C, H, W, K);
          conv_forward_kernel_layer2<<<DimGrid, DimBlock, 0, stream1>>> (device_y1, device_x1, B, M, C, H, W, K); 
        }

        hipMemcpyAsync(host_pinned_y + i * (y_len_stream/small_B), device_y0, y_len_stream * sizeof(float), hipMemcpyDeviceToHost, stream0);
        hipMemcpyAsync(host_pinned_y + i * (y_len_stream/small_B) + y_len_stream, device_y1, y_len_stream * sizeof(float), hipMemcpyDeviceToHost, stream1);
    
        hipStreamSynchronize(stream1);
      }

    #pragma unroll NUM_PTHREADS
    for (unsigned int i = 0; i <  NUM_PTHREADS; i++) {
        args_pinned_list[i].len = y_len;
        args_pinned_list[i].copy_from_address = host_pinned_y;
        args_pinned_list[i].copy_to_address = host_y;
        args_pinned_list[i].start_idx = i*ceil((1.0*y_len)/NUM_PTHREADS);
        args_pinned_list[i].end_idx = (i+1)*ceil((1.0*y_len)/NUM_PTHREADS);
        pthread_create(tids + i, NULL, pinned_transfer, (void *)(args_pinned_list + i));
    }

    #pragma unroll NUM_PTHREADS
    for (unsigned int i = 0; i <  NUM_PTHREADS; i++) {
      pthread_join(tids[i], NULL);
    }

    hipFree(device_y0);
    hipFree(device_y1);
    hipFree(device_x0);
    hipFree(device_x1);
    hipHostFree(host_pinned_x);
    hipHostFree(host_pinned_y);

    free(host_k16);
}

__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
        std::cout<<"Device Overlap: "<<deviceProp.deviceOverlap<<std::endl;
    }
}
